#include "hip/hip_runtime.h"
#define GGML_ASSERT(...)
#define GGML_ABORT(...)
#define GGML_UNUSED(x) (void)(x)

template <size_t split_d_inner, size_t d_conv>
static __global__ void ssm_conv_f32(const float* __restrict__ src0, const float* __restrict__ src1,
    const int src0_nb0, const int src0_nb1, const int src0_nb2, const int src1_nb1,
    float* __restrict__ dst, const int dst_nb0, const int dst_nb1, const int dst_nb2,
    const int64_t n_t) {
    GGML_UNUSED(src0_nb0);
    const int tid = threadIdx.x;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;

    const float* x_block = (const float*)((const char*)src0 + bidx * src0_nb2 + bidy * split_d_inner * src0_nb1);
    const float* w_block = (const float*)((const char*)src1 + bidy * split_d_inner * src1_nb1);
    float* y_block = (float*)((char*)dst + bidx * dst_nb2 + bidy * split_d_inner * dst_nb0);

    const int stride_x = src0_nb1 / sizeof(float);
    const int stride_w = src1_nb1 / sizeof(float);
    const int stride_y = dst_nb1 / sizeof(float);

    float x[d_conv] = { 0.0f };
    float w[d_conv] = { 0.0f };

#pragma unroll
    for (size_t j = 0; j < d_conv; j++) {
        w[j] = w_block[tid * stride_w + j];
    }

    for (int64_t i = 0; i < n_t; i++) {
        float sumf = 0.0f;

        if (i == 0) {
            for (size_t j = 0; j < d_conv; j++) {
                x[j] = x_block[tid * stride_x + j];
            }
        }
        else {
            x[(i - 1) % d_conv] = x_block[tid * stride_x + i + d_conv - 1];
        }

#pragma unroll
        for (size_t j = 0; j < d_conv; j++) {
            sumf += x[(i + j) % d_conv] * w[j];
        }
        y_block[i * stride_y + tid] = sumf;
    }
}

template <size_t split_d_inner, size_t d_conv, int64_t split_n_t>
static __global__ void ssm_conv_long_token_f32(const float* __restrict__ src0, const float* __restrict__ src1,
    const int src0_nb0, const int src0_nb1, const int src0_nb2,
    const int src1_nb1, float* __restrict__ dst, const int dst_nb0,
    const int dst_nb1, const int dst_nb2, const int64_t n_t) {
    const int tid = threadIdx.x;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;
    const int bidz = blockIdx.z;

    const float* x_block = (const float*)((const char*)src0 + bidx * src0_nb2 + bidy * split_d_inner * src0_nb1 +
        bidz * split_n_t * src0_nb0);
    const float* w_block = (const float*)((const char*)src1 + bidy * split_d_inner * src1_nb1);
    float* y_block =
        (float*)((char*)dst + bidx * dst_nb2 + bidz * split_n_t * dst_nb1 + bidy * split_d_inner * dst_nb0);

    const int stride_x = src0_nb1 / sizeof(float);
    const int stride_w = src1_nb1 / sizeof(float);
    const int stride_y = dst_nb1 / sizeof(float);

    float x[d_conv] = { 0.0f };
    float w[d_conv] = { 0.0f };

#pragma unroll
    for (size_t j = 0; j < d_conv; j++) {
        w[j] = w_block[tid * stride_w + j];
    }

#pragma unroll
    for (int64_t i = 0; i < split_n_t; i++) {
        if (bidz * split_n_t + i < n_t) {
            float sumf = 0.0f;

            if (i == 0) {
                for (size_t j = 0; j < d_conv; j++) {
                    x[j] = x_block[tid * stride_x + j];
                }
            }
            else {
                x[(i - 1) % d_conv] = x_block[tid * stride_x + i + d_conv - 1];
            }

#pragma unroll
            for (size_t j = 0; j < d_conv; j++) {
                sumf += x[(i + j) % d_conv] * w[j];
            }
            y_block[i * stride_y + tid] = sumf;
        }
    }
}

void ssm_conv_f32_cuda(const float* src0, const float* src1, const int src0_nb0, const int src0_nb1,
    const int src0_nb2, const int src1_nb1, float* dst, const int dst_nb0, const int dst_nb1,
    const int dst_nb2, const int64_t nc, const int64_t nr, const int64_t n_t,
    const int64_t n_s, hipStream_t stream) {
    const int threads = 128;
    GGML_ASSERT(nr % threads == 0);

    if (n_t <= 32) {
        const dim3 blocks(n_s, (nr + threads - 1) / threads, 1);
        if (nc == 4) {
            ssm_conv_f32<threads, 4> << <blocks, threads, 0, stream >> > (src0, src1, src0_nb0, src0_nb1, src0_nb2, src1_nb1,
                dst, dst_nb0, dst_nb1, dst_nb2, n_t);
        }
        else if (nc == 3) {
            ssm_conv_f32<threads, 3> << <blocks, threads, 0, stream >> > (src0, src1, src0_nb0, src0_nb1, src0_nb2, src1_nb1,
                dst, dst_nb0, dst_nb1, dst_nb2, n_t);
        }
        else {
            GGML_ABORT("Only support kernel size = 3 or size = 4 right now.");
        }
    }
    else {
        if (nc == 4) {
            const int64_t split_n_t = 32;
            dim3          blocks(n_s, (nr + threads - 1) / threads, (n_t + split_n_t - 1) / split_n_t);
            ssm_conv_long_token_f32<threads, 4, split_n_t> << <blocks, threads, 0, stream >> > (
                src0, src1, src0_nb0, src0_nb1, src0_nb2, src1_nb1, dst, dst_nb0, dst_nb1, dst_nb2, n_t);
        }
        else if (nc == 3) {
            const int64_t split_n_t = 32;
            dim3          blocks(n_s, (nr + threads - 1) / threads, (n_t + split_n_t - 1) / split_n_t);
            ssm_conv_long_token_f32<threads, 3, split_n_t> << <blocks, threads, 0, stream >> > (
                src0, src1, src0_nb0, src0_nb1, src0_nb2, src1_nb1, dst, dst_nb0, dst_nb1, dst_nb2, n_t);
        }
        else {
            GGML_ABORT("Only support kernel size = 3 or size = 4 right now.");
        }
    }
}