#include "hip/hip_runtime.h"
#include "cuda_func.h"
#include "common.cuh"
#define GGML_ASSERT(...)

static __device__ __forceinline__ float op_repeat(const float, const float b) {
    return b;
}

static __device__ __forceinline__ float op_add(const float a, const float b) {
    return a + b;
}

static __device__ __forceinline__ float op_sub(const float a, const float b) {
    return a - b;
}

static __device__ __forceinline__ float op_mul(const float a, const float b) {
    return a * b;
}

static __device__ __forceinline__ float op_div(const float a, const float b) {
    return a / b;
}

template <float (*bin_op)(const float, const float),
    typename src0_t,
    typename src1_t,
    typename dst_t,
    typename... src1_ptrs>
static __global__ void k_bin_bcast_unravel(const src0_t* src0,
    const src1_t* src1,
    dst_t* dst,
    const uint3            ne0,
    const uint3            ne1,
    const uint3            ne2,
    const uint32_t         ne3,
    const uint3            prod_012,
    const uint3            prod_01,
    const uint3            ne10,
    const uint3            ne11,
    const uint3            ne12,
    const uint3            ne13,
    /*int s0, */ const int s1,
    const int              s2,
    const int              s3,
    /*int s00,*/ const int s01,
    const int              s02,
    const int              s03,
    /*int s10,*/ const int s11,
    const int              s12,
    const int              s13,
    src1_ptrs... src1s) {
    const int i = blockDim.x * blockIdx.x + threadIdx.x;

    const uint32_t i3 = fastdiv(i, prod_012);
    const uint32_t i2 = fastdiv(i - i3 * prod_012.z, prod_01);
    const uint32_t i1 = fastdiv(i - i3 * prod_012.z - i2 * prod_01.z, ne0);
    const uint32_t i0 = i - i3 * prod_012.z - i2 * prod_01.z - i1 * ne0.z;

    if (i0 >= ne0.z || i1 >= ne1.z || i2 >= ne2.z || i3 >= ne3) {
        return;
    }

    const int i11 = fastmodulo(i1, ne11);
    const int i12 = fastmodulo(i2, ne12);
    const int i13 = fastmodulo(i3, ne13);

    const size_t i_src0 = i3 * s03 + i2 * s02 + i1 * s01;
    const size_t i_src1 = i13 * s13 + i12 * s12 + i11 * s11;
    const size_t i_dst = i3 * s3 + i2 * s2 + i1 * s1;

    const src0_t* src0_row = src0 ? (src0 + i_src0) : nullptr;
    dst_t* dst_row = dst + i_dst;

    const int i10 = fastmodulo(i0, ne10);

    float result = src0_row ? (float)src0_row[i0] : 0.0f;
    if constexpr (sizeof...(src1_ptrs) > 0) {
        result = (..., (result = bin_op(result, (float)src1s[i_src1 + i10])));
    }
    else {
        result = bin_op(result, (float)src1[i_src1 + i10]);
    }

    dst_row[i0] = (dst_t)result;
}

template <float (*bin_op)(const float, const float),
    typename src0_t,
    typename src1_t,
    typename dst_t,
    typename... src1_ptrs>
static __global__ void k_bin_bcast(const src0_t* src0,
    const src1_t* src1,
    dst_t* dst,
    const int              ne0,
    const int              ne1,
    const int              ne2,
    const uint3            ne3,
    const uint3            ne10,
    const uint3            ne11,
    const uint3            ne12,
    const uint3            ne13,
    /*int s0, */ const int s1,
    const int              s2,
    const int              s3,
    /*int s00,*/ const int s01,
    const int              s02,
    const int              s03,
    /*int s10,*/ const int s11,
    const int              s12,
    const int              s13,
    src1_ptrs... src1s) {
    const uint32_t i0s = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t i1 = (blockDim.y * blockIdx.y + threadIdx.y);
    const uint32_t i2 = fastdiv((blockDim.z * blockIdx.z + threadIdx.z), ne3);
    const uint32_t i3 = (blockDim.z * blockIdx.z + threadIdx.z) - (i2 * ne3.z);

    if (i0s >= (uint32_t)ne0 || i1 >= (uint32_t)ne1 || i2 >= (uint32_t)ne2 || i3 >= ne3.z) {
        return;
    }

    const uint32_t i11 = fastmodulo(i1, ne11);
    const uint32_t i12 = fastmodulo(i2, ne12);
    const uint32_t i13 = fastmodulo(i3, ne13);

    const size_t i_src0 = i3 * s03 + i2 * s02 + i1 * s01;
    const size_t i_src1 = i13 * s13 + i12 * s12 + i11 * s11;
    const size_t i_dst = i3 * s3 + i2 * s2 + i1 * s1;

    const src0_t* src0_row = src0 ? (src0 + i_src0) : nullptr;
    dst_t* dst_row = dst + i_dst;

    for (int i0 = i0s; i0 < ne0; i0 += blockDim.x * gridDim.x) {
        const uint32_t i10 = fastmodulo(i0, ne10);

        float result = src0_row ? (float)src0_row[i0] : 0.0f;
        if constexpr (sizeof...(src1_ptrs) > 0) {
            result = (..., (result = bin_op(result, (float)src1s[i_src1 + i10])));
        }
        else {
            result = bin_op(result, (float)src1[i_src1 + i10]);
        }

        dst_row[i0] = (dst_t)result;
    }
}

template<class op>
static void ggml_cuda_op_bin_bcast(const bin_bcast_context* ctx, hipStream_t stream) {
    GGML_ASSERT(src1->type == GGML_TYPE_F32 || src1->type == GGML_TYPE_F16);
    if (ctx->src0_type == GGML_TYPE_F32 && ctx->dst_type == GGML_TYPE_F32) {
        op().template operator() < float, float, float > (ctx, stream);
    }
    else if (ctx->src0_type == GGML_TYPE_F16 && ctx->src1_type == GGML_TYPE_F16 && ctx->dst_type == GGML_TYPE_F16) {
        op().template operator() < half, half, half > (ctx, stream);
    }
    else if (ctx->src0_type == GGML_TYPE_F16 && ctx->src1_type == GGML_TYPE_F32 && ctx->dst_type == GGML_TYPE_F16) {
        op().template operator() < half, float, half > (ctx, stream);
    }
    else if (ctx->src0_type == GGML_TYPE_F16 && ctx->dst_type == GGML_TYPE_F32) {
        op().template operator() < half, float, float > (ctx, stream);
    }
    else if (ctx->src0_type == GGML_TYPE_BF16 && ctx->src1_type == GGML_TYPE_BF16) {
        op().template operator() < hip_bfloat16, hip_bfloat16, hip_bfloat16 > (ctx, stream);
    }
    else if (ctx->src0_type == GGML_TYPE_BF16 && ctx->src1_type == GGML_TYPE_F32) {
        op().template operator() < hip_bfloat16, float, hip_bfloat16 > (ctx, stream);
    }
    else {
        //fprintf(stderr, "%s: unsupported types: dst: %s, src0: %s, src1: %s\n", __func__,
            //ggml_type_name(dst->type), ggml_type_name(src0->type), ggml_type_name(src1->type));
        GGML_ABORT("fatal error");
    }
}

template <float (*bin_op)(const float, const float), typename src0_t, typename src1_t, typename dst_t, size_t... I>
static void launch_bin_bcast_pack(const bin_bcast_context* ctx, hipStream_t stream, std::index_sequence<I...>) {

    int nr0 = ctx->ne10 / ctx->ne0;
    int nr1 = ctx->ne11 / ctx->ne1;
    int nr2 = ctx->ne12 / ctx->ne2;
    int nr3 = ctx->ne13 / ctx->ne3;

    int nr[4] = { nr0, nr1, nr2, nr3 };

    int64_t cne[] = { ctx->ne0, ctx->ne1, ctx->ne2, ctx->ne3 };
    int64_t cne0[] = { ctx->ne00, ctx->ne01, ctx->ne02, ctx->ne03 };
    int64_t cne1[] = { ctx->ne10, ctx->ne11, ctx->ne12, ctx->ne13 };

    size_t cnb[] = { ctx->nb0, ctx->nb1, ctx->nb2, ctx->nb3 };
    size_t cnb0[] = { ctx->nb00, ctx->nb01, ctx->nb02, ctx->nb03 };
    size_t cnb1[] = { ctx->nb10, ctx->nb11, ctx->nb12, ctx->nb13 };

    auto collapse = [](int64_t cne[]) {
        cne[0] *= cne[1];
        cne[1] = cne[2];
        cne[2] = cne[3];
        cne[3] = 1;
    };

    auto collapse_nb = [](size_t cnb[], const int64_t cne[]) {
        cnb[1] *= cne[1];
        cnb[2] *= cne[2];
        cnb[3] *= cne[3];
    };

    if (ctx->src0_is_contiguous && ctx->src1_is_contiguous && ctx->dst_is_contiguous) {
        for (int i = 0; i < 4; i++) {
            if (nr[i] != 1) {
                break;
            }
            if (i > 0) {
                collapse_nb(cnb, cne);
                collapse_nb(cnb0, cne0);
                collapse_nb(cnb1, cne1);
                collapse(cne);
                collapse(cne0);
                collapse(cne1);
            }
        }
    }

    {
        int64_t ne0 = cne[0];
        int64_t ne1 = cne[1];
        int64_t ne2 = cne[2];
        int64_t ne3 = cne[3];

        //[[maybe_unused]] int64_t ne00 = cne0[0];
        //[[maybe_unused]] int64_t ne01 = cne0[1];
        //[[maybe_unused]] int64_t ne02 = cne0[2];
        //[[maybe_unused]] int64_t ne03 = cne0[3];

        size_t nb0 = cnb[0];
        size_t nb1 = cnb[1];
        size_t nb2 = cnb[2];
        size_t nb3 = cnb[3];

        size_t nb00 = cnb0[0];
        size_t nb01 = cnb0[1];
        size_t nb02 = cnb0[2];
        size_t nb03 = cnb0[3];

        size_t nb10 = cnb1[0];
        size_t nb11 = cnb1[1];
        size_t nb12 = cnb1[2];
        size_t nb13 = cnb1[3];

        size_t s0 = nb0 / sizeof(dst_t);
        size_t s1 = nb1 / sizeof(dst_t);
        size_t s2 = nb2 / sizeof(dst_t);
        size_t s3 = nb3 / sizeof(dst_t);

        size_t s10 = nb10 / sizeof(src1_t);
        size_t s11 = nb11 / sizeof(src1_t);
        size_t s12 = nb12 / sizeof(src1_t);
        size_t s13 = nb13 / sizeof(src1_t);

        size_t s00 = nb00 / sizeof(src0_t);
        size_t s01 = nb01 / sizeof(src0_t);
        size_t s02 = nb02 / sizeof(src0_t);
        size_t s03 = nb03 / sizeof(src0_t);

        GGML_ASSERT(nb0 % sizeof(dst_t) == 0);
        GGML_ASSERT(nb1 % sizeof(dst_t) == 0);
        GGML_ASSERT(nb2 % sizeof(dst_t) == 0);
        GGML_ASSERT(nb3 % sizeof(dst_t) == 0);

        GGML_ASSERT(nb00 % sizeof(src0_t) == 0);
        GGML_ASSERT(nb01 % sizeof(src0_t) == 0);
        GGML_ASSERT(nb02 % sizeof(src0_t) == 0);
        GGML_ASSERT(nb03 % sizeof(src0_t) == 0);

        GGML_ASSERT(nb10 % sizeof(src1_t) == 0);
        GGML_ASSERT(nb11 % sizeof(src1_t) == 0);
        GGML_ASSERT(nb12 % sizeof(src1_t) == 0);
        GGML_ASSERT(nb13 % sizeof(src1_t) == 0);

        GGML_ASSERT(s0 == 1);
        GGML_ASSERT(s00 == 1);
        GGML_ASSERT(s10 == 1);

        const int block_size = 128;

        int64_t hne0 = std::max(ne0 / 2LL, 1LL);

        dim3 block_dims;
        block_dims.x = std::min<unsigned int>(hne0, block_size);
        block_dims.y = std::min<unsigned int>(ne1, block_size / block_dims.x);
        block_dims.z = std::min(std::min<unsigned int>(ne2 * ne3, block_size / block_dims.x / block_dims.y), 64U);

        dim3 block_nums((hne0 + block_dims.x - 1) / block_dims.x, (ne1 + block_dims.y - 1) / block_dims.y,
            (ne2* ne3 + block_dims.z - 1) / block_dims.z);

        const uint3 ne10 = init_fastdiv_values((uint32_t)cne1[0]);
        const uint3 ne11 = init_fastdiv_values((uint32_t)cne1[1]);
        const uint3 ne12 = init_fastdiv_values((uint32_t)cne1[2]);
        const uint3 ne13 = init_fastdiv_values((uint32_t)cne1[3]);

        if (block_nums.z > 65535) {
            int         block_num = (ne0 * ne1 * ne2 * ne3 + block_size - 1) / block_size;
            const uint3 prod_012 = init_fastdiv_values((uint32_t)(ne0 * ne1 * ne2));
            const uint3 prod_01 = init_fastdiv_values((uint32_t)(ne0 * ne1));
            const uint3 ne0_fastdiv = init_fastdiv_values((uint32_t)ne0);
            const uint3 ne1_fastdiv = init_fastdiv_values((uint32_t)ne1);
            const uint3 ne2_fastdiv = init_fastdiv_values((uint32_t)ne2);

            if constexpr (sizeof...(I) > 0) {
                k_bin_bcast_unravel<bin_op, src0_t, src1_t, dst_t> << <block_num, block_size, 0, stream >> > (
                    (const src0_t*)ctx->src_data[0], (const src1_t*)ctx->src_data[1], (dst_t*)ctx->dst_d, ne0_fastdiv, ne1_fastdiv, ne2_fastdiv, ne3, prod_012, prod_01, ne10, ne11,
                    ne12, ne13,
                    /* s0, */ s1, s2, s3,
                    /* s00,*/ s01, s02, s03,
                    /* s10,*/ s11, s12, s13, (const src1_t*)ctx->src_data[I + 1]...);
            }
            else {
                k_bin_bcast_unravel<bin_op, src0_t, src1_t, dst_t>
                    << <block_num, block_size, 0, stream >> > ((const src0_t*)ctx->src_data[0], (const src1_t*)ctx->src_data[1], (dst_t*)ctx->dst_d, ne0_fastdiv, ne1_fastdiv,
                        ne2_fastdiv, ne3, prod_012, prod_01, ne10, ne11, ne12, ne13,
                        /* s0, */ s1, s2, s3,
                        /* s00,*/ s01, s02, s03,
                        /* s10,*/ s11, s12, s13);
            }
        }
        else {
            const uint3 ne3_fastdiv = init_fastdiv_values((uint32_t)ne3);
            if constexpr (sizeof...(I) > 0) {
                k_bin_bcast<bin_op, src0_t, src1_t, dst_t> << <block_nums, block_dims, 0, stream >> > (
                    (const src0_t*)ctx->src_data[0], (const src1_t*)ctx->src_data[1], (dst_t*)ctx->dst_d, ne0, ne1, ne2, ne3_fastdiv, ne10, ne11, ne12, ne13,
                    /* s0, */ s1, s2, s3,
                    /* s00,*/ s01, s02, s03,
                    /* s10,*/ s11, s12, s13, (const src1_t*)ctx->src_data[I + 1]...);
            }
            else {
                k_bin_bcast<bin_op, src0_t, src1_t, dst_t> << <block_nums, block_dims, 0, stream >> > (
                    (const src0_t*)ctx->src_data[0], (const src1_t*)ctx->src_data[1], (dst_t*)ctx->dst_d, ne0, ne1, ne2, ne3_fastdiv, ne10, ne11, ne12, ne13,
                    /* s0, */ s1, s2, s3,
                    /* s00,*/ s01, s02, s03,
                    /* s10,*/ s11, s12, s13);
            }
        }
    }
}

template <float (*bin_op)(const float, const float), int n_fuse = 1>
struct bin_bcast_cuda {
    template<typename src0_t, typename src1_t, typename dst_t>
    void operator()(const bin_bcast_context* ctx, hipStream_t stream) {
        launch_bin_bcast_pack<bin_op, src0_t, src1_t, dst_t>(
            ctx, stream, std::make_index_sequence<n_fuse>{});
    }
};

void repeat_cuda(const bin_bcast_context* ctx, hipStream_t stream)
{
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_repeat, 0>>(ctx, stream);
}

void add_cuda(const bin_bcast_context* ctx, hipStream_t stream)
{
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_add>>(ctx, stream);
}

void sub_cuda(const bin_bcast_context* ctx, hipStream_t stream)
{
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_sub>>(ctx, stream);
}

void mul_cuda(const bin_bcast_context* ctx, hipStream_t stream)
{
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_mul>>(ctx, stream);
}

void div_cuda(const bin_bcast_context* ctx, hipStream_t stream)
{
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_div>>(ctx, stream);
}

template <float (*op)(const float, const float), int n_fuse>
static void ggml_cuda_op_fused_binbcast_impl(const bin_bcast_context* ctx, hipStream_t stream) {
    if (ctx->src0_type == GGML_TYPE_F32 && ctx->dst_type == GGML_TYPE_F32) {
        launch_bin_bcast_pack<op, float, float, float>(ctx, stream, std::make_index_sequence<n_fuse>{});
    }
    else if (ctx->src0_type == GGML_TYPE_F16 && ctx->src1_type == GGML_TYPE_F16 && ctx->dst_type == GGML_TYPE_F16) {
        launch_bin_bcast_pack<op, half, half, half>(ctx, stream, std::make_index_sequence<n_fuse>{});
    }
    else if (ctx->src0_type == GGML_TYPE_F16 && ctx->src1_type == GGML_TYPE_F32 && ctx->dst_type == GGML_TYPE_F16) {
        launch_bin_bcast_pack<op, half, float, half>(ctx, stream, std::make_index_sequence<n_fuse>{});
    }
    else if (ctx->src0_type == GGML_TYPE_F16 && ctx->dst_type == GGML_TYPE_F32) {
        launch_bin_bcast_pack<op, half, float, float>(ctx, stream, std::make_index_sequence<n_fuse>{});
    }
    else {
#if 0
        fprintf(stderr,
            "%s: unsupported types for fusion: dst: %s, src0: %s, src1: %s\n",
            __func__, ggml_type_name(ctx->dst_type), ggml_type_name(ctx->src0_type), ggml_type_name(ctx->src1_type));
        GGML_ABORT("fatal error");
#endif
    }
}

template <typename T>
static __global__ void k_repeat_back(
    const T* __restrict__ src, T* __restrict__ dst, const int64_t ne00, const int64_t ne01, const int64_t ne02, const int64_t ne03,
    const size_t s00, const size_t s01, const size_t s02, const size_t s03,
    const int64_t ne0, const int64_t ne1, const int64_t ne2, const int64_t ne3) {

    const int64_t tid0 = int64_t(blockIdx.x) * blockDim.x + threadIdx.x;
    const int64_t tid1 = int64_t(blockIdx.y) * blockDim.y + threadIdx.y;
    const int64_t tid23 = int64_t(blockIdx.z) * blockDim.z + threadIdx.z;
    const int64_t tid2 = tid23 % ne2;
    const int64_t tid3 = tid23 / ne2;

    if (tid0 >= ne0) {
        return;
    }

    T sum = 0;
    for (int64_t i3 = tid3; i3 < ne03; i3 += ne3) {
        for (int64_t i2 = tid2; i2 < ne02; i2 += ne2) {
            for (int64_t i1 = tid1; i1 < ne01; i1 += ne1) {
                for (int64_t i0 = tid0; i0 < ne00; i0 += ne0) {
                    sum += src[i3 * s03 + i2 * s02 + i1 * s01 + i0 * s00];
                }
            }
        }
    }
    dst[tid3 * ne2 * ne1 * ne0 + tid2 * ne1 * ne0 + tid1 * ne0 + tid0] = sum;
}

template <typename T>
static void repeat_back_cuda(
    const T* src, T* dst, const int64_t ne00, const int64_t ne01, const int64_t ne02, const int64_t ne03,
    const size_t s00, const size_t s01, const size_t s02, const size_t s03,
    const int64_t ne0, const int64_t ne1, const int64_t ne2, const int64_t ne3, hipStream_t stream) {

    const dim3 block_dims(WARP_SIZE, 1, 1);
    const dim3 block_nums((ne0 + WARP_SIZE - 1) / WARP_SIZE, ne1, ne2 * ne3);
    k_repeat_back<T> << <block_nums, block_dims, 0, stream >> >
        (src, dst, ne00, ne01, ne02, ne03, s00, s01, s02, s03, ne0, ne1, ne2, ne3);
}

void repeat_back_cuda(const repeat_back_context* ctx, hipStream_t stream)
{
    const size_t s00 = ctx->nb00 / ctx->src0_ts;
    const size_t s01 = ctx->nb01 / ctx->src0_ts;
    const size_t s02 = ctx->nb02 / ctx->src0_ts;
    const size_t s03 = ctx->nb03 / ctx->src0_ts;
    switch (ctx->dst_type) {
    case GGML_TYPE_F32: {
        const float* src0_d = (const float*)ctx->src0_d;
        float* dst_d = (float*)ctx->dst_d;
        repeat_back_cuda(src0_d, dst_d,
            ctx->ne00, ctx->ne01, ctx->ne02, ctx->ne03,
            s00, s01, s02, s03,
            ctx->ne0, ctx->ne1, ctx->ne2, ctx->ne3, stream);
    } break;
    default: {
        GGML_ASSERT(false);
    } break;
    }
}

void fused_add_cuda(const bin_bcast_context* ctx, int n_fuse, hipStream_t stream) {
    GGML_ASSERT(2 <= n_fuse && n_fuse <= 8);

    switch (n_fuse) {
    case 2:
        ggml_cuda_op_fused_binbcast_impl<op_add, 2>(ctx, stream);
        break;
    case 3:
        ggml_cuda_op_fused_binbcast_impl<op_add, 3>(ctx, stream);
        break;
    case 4:
        ggml_cuda_op_fused_binbcast_impl<op_add, 4>(ctx, stream);
        break;
    case 5:
        ggml_cuda_op_fused_binbcast_impl<op_add, 5>(ctx, stream);
        break;
    case 6:
        ggml_cuda_op_fused_binbcast_impl<op_add, 6>(ctx, stream);
        break;
    case 7:
        ggml_cuda_op_fused_binbcast_impl<op_add, 7>(ctx, stream);
        break;
    case 8:
        ggml_cuda_op_fused_binbcast_impl<op_add, 8>(ctx, stream);
        break;
    default:
        GGML_ASSERT(false && "Unsupported n_fuse value");
    }
}