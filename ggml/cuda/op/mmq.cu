#include "hip/hip_runtime.h"
#include <assert.h>
#include "cuda_func.h"
#include "internal_ds.h"
#include "mmq.cuh"

// To reduce shared memory use, store "it" and "iex_used" with 22/10 bits each.
struct mmq_ids_helper_store {
    uint32_t data;

    __device__ mmq_ids_helper_store(const uint32_t it, const uint32_t iex_used) {
        data = (it & 0x003FFFFF) | (iex_used << 22);
    }

    __device__ uint32_t it() const {
        return data & 0x003FFFFF;
    }

    __device__ uint32_t iex_used() const {
        return data >> 22;
    }
};
static_assert(sizeof(mmq_ids_helper_store) == 4, "unexpected size for mmq_ids_helper_store");

// Helper function for mul_mat_id, converts ids to a more convenient format.
// ids_src1 describes how to permute the flattened column indices of src1 in order to get a compact src1 tensor sorted by expert.
// ids_dst describes the same mapping but for the dst tensor.
// The upper and lower bounds for the ith expert in the compact src1 tensor are stored in expert_bounds[i:i+1].
template <int n_expert_used_template>
__launch_bounds__(ggml_cuda_get_physical_warp_size(), 1)
static __global__ void mmq_ids_helper(
    const int32_t* __restrict__ ids, int32_t* __restrict__ ids_src1, int32_t* __restrict__ ids_dst, int32_t* __restrict__ expert_bounds,
    const int n_tokens, const int n_expert_used_var, const int nchannels_y, const int si1, const int sis1) {
    constexpr int warp_size = ggml_cuda_get_physical_warp_size();
    const int n_expert_used = n_expert_used_template == 0 ? n_expert_used_var : n_expert_used_template;
    const int expert = blockIdx.x;

    extern __shared__ char data_mmq_ids_helper[];
    mmq_ids_helper_store* store = (mmq_ids_helper_store*)data_mmq_ids_helper;

    int nex_prev = 0; // Number of columns for experts with a lower index.
    int it_compact = 0; // Running index for the compact slice of this expert.

    if constexpr (n_expert_used_template == 0) {
        // Generic implementation:
        for (int it = 0; it < n_tokens; ++it) {
            int iex_used = -1; // The index at which the expert is used, if any.
            for (int iex = threadIdx.x; iex < n_expert_used; iex += warp_size) {
                const int expert_used = ids[it * si1 + iex];
                nex_prev += expert_used < expert;
                if (expert_used == expert) {
                    iex_used = iex;
                }
            }

            if (iex_used != -1) {
                store[it_compact] = mmq_ids_helper_store(it, iex_used);
            }

            if (warp_reduce_any<warp_size>(iex_used != -1)) {
                it_compact++;
            }
        }
    }
    else {
        // Implementation optimized for specific numbers of experts used:
        static_assert(n_expert_used == 6 || warp_size % n_expert_used == 0, "bad n_expert_used");
        const int neu_padded = n_expert_used == 6 ? 8 : n_expert_used; // Padded to next higher power of 2.
        for (int it0 = 0; it0 < n_tokens; it0 += warp_size / neu_padded) {
            const int it = it0 + threadIdx.x / neu_padded;

            const int iex = threadIdx.x % neu_padded; // The index at which the expert is used, if any.
            const int expert_used = (neu_padded == n_expert_used || iex < n_expert_used) && it < n_tokens ?
                ids[it * si1 + iex] : INT_MAX;
            const int iex_used = expert_used == expert ? iex : -1;
            nex_prev += expert_used < expert;

            // Whether the threads at this token position have used the expert:
            const int it_compact_add_self = warp_reduce_any<neu_padded>(iex_used != -1);

            // Do a scan over threads at lower token positions in warp to get the correct index for writing data:
            int it_compact_add_lower = 0;
#pragma unroll
            for (int offset = neu_padded; offset < warp_size; offset += neu_padded) {
                const int tmp = __shfl_up_sync(0xFFFFFFFF, it_compact_add_self, offset, warp_size);
                if (threadIdx.x >= static_cast<unsigned int>(offset)) {
                    it_compact_add_lower += tmp;
                }
            }

            if (iex_used != -1) {
                store[it_compact + it_compact_add_lower] = mmq_ids_helper_store(it, iex_used);
            }

            // The thread with the highest index in the warp always has the sum over the whole warp, use it to increment all threads:
            it_compact += __shfl_sync(0xFFFFFFFF, it_compact_add_lower + it_compact_add_self, warp_size - 1, warp_size);
        }
    }
    nex_prev = warp_reduce_sum<warp_size>(nex_prev);

    for (int itc = threadIdx.x; itc < it_compact; itc += warp_size) {
        const mmq_ids_helper_store store_it = store[itc];
        const int it = store_it.it();
        const int iex_used = store_it.iex_used();
        ids_src1[nex_prev + itc] = it * sis1 + iex_used % nchannels_y;
        ids_dst[nex_prev + itc] = it * n_expert_used + iex_used;
    }

    if (threadIdx.x != 0) {
        return;
    }

    expert_bounds[expert] = nex_prev;

    if (expert < static_cast<int>(gridDim.x) - 1) {
        return;
    }

    expert_bounds[gridDim.x] = nex_prev + it_compact;
}

template <int n_expert_used_template>
static void launch_mmq_ids_helper(
    const int32_t* __restrict__ ids, int32_t* __restrict__ ids_src1, int32_t* __restrict__ ids_dst, int32_t* __restrict__ expert_bounds,
    const int n_experts, const int n_tokens, const int n_expert_used_var, const int nchannels_y, const int si1, const int sis1, hipStream_t stream) {
    GGML_ASSERT(n_tokens < (1 << 22) && "too few bits in mmq_ids_helper_store");
    GGML_ASSERT(n_expert_used_var < (1 << 10) && "too few bits in mmq_ids_helper_store");

    const int id = ggml_cuda_get_device();
    const int warp_size = ggml_cuda_info().devices[id].warp_size;
    const size_t smpbo = ggml_cuda_info().devices[id].smpbo;
    CUDA_SET_SHARED_MEMORY_LIMIT(reinterpret_cast<const void*>(mmq_ids_helper<n_expert_used_template>), smpbo);

    const dim3 num_blocks(n_experts, 1, 1);
    const dim3 block_size(warp_size, 1, 1);
    const size_t nbytes_shared = n_tokens * sizeof(mmq_ids_helper_store);
    GGML_ASSERT(nbytes_shared <= smpbo);
    mmq_ids_helper<n_expert_used_template> << <num_blocks, block_size, nbytes_shared, stream >> >
        (ids, ids_src1, ids_dst, expert_bounds, n_tokens, n_expert_used_var, nchannels_y, si1, sis1);
}

void launch_mmq_ids_helper(const mmq_ids_helper_context* ctx, hipStream_t stream)
{
    switch (ctx->n_expert_used) {
    case  2:
        launch_mmq_ids_helper< 2>(ctx->ids, ctx->ids_src1, ctx->ids_dst, ctx->expert_bounds,
            ctx->n_experts, ctx->n_tokens, ctx->n_expert_used, ctx->nchannels_y, ctx->si1, ctx->sis1, stream);
        break;
    case  4:
        launch_mmq_ids_helper< 4>(ctx->ids, ctx->ids_src1, ctx->ids_dst, ctx->expert_bounds,
            ctx->n_experts, ctx->n_tokens, ctx->n_expert_used, ctx->nchannels_y, ctx->si1, ctx->sis1, stream);
        break;
    case  6:
        launch_mmq_ids_helper< 6>(ctx->ids, ctx->ids_src1, ctx->ids_dst, ctx->expert_bounds,
            ctx->n_experts, ctx->n_tokens, ctx->n_expert_used, ctx->nchannels_y, ctx->si1, ctx->sis1, stream);
        break;
    case  8:
        launch_mmq_ids_helper< 8>(ctx->ids, ctx->ids_src1, ctx->ids_dst, ctx->expert_bounds,
            ctx->n_experts, ctx->n_tokens, ctx->n_expert_used, ctx->nchannels_y, ctx->si1, ctx->sis1, stream);
        break;
    case 16:
        launch_mmq_ids_helper<16>(ctx->ids, ctx->ids_src1, ctx->ids_dst, ctx->expert_bounds,
            ctx->n_experts, ctx->n_tokens, ctx->n_expert_used, ctx->nchannels_y, ctx->si1, ctx->sis1, stream);
        break;
    case 32:
        launch_mmq_ids_helper<32>(ctx->ids, ctx->ids_src1, ctx->ids_dst, ctx->expert_bounds,
            ctx->n_experts, ctx->n_tokens, ctx->n_expert_used, ctx->nchannels_y, ctx->si1, ctx->sis1, stream);
        break;
    default:
        launch_mmq_ids_helper< 0>(ctx->ids, ctx->ids_src1, ctx->ids_dst, ctx->expert_bounds,
            ctx->n_experts, ctx->n_tokens, ctx->n_expert_used, ctx->nchannels_y, ctx->si1, ctx->sis1, stream);
        break;
    }
    CUDA_CHECK(hipGetLastError());
}

void ggml_cuda_mul_mat_q_switch_type(ggml_cuda_pool& pool, const mmq_args& args, hipStream_t stream) {
    switch (args.type_x) {
    case GGML_TYPE_Q4_0:
        mul_mat_q_case<GGML_TYPE_Q4_0, block_q4_0>(pool, args, stream);
        break;
    case GGML_TYPE_Q4_1:
        mul_mat_q_case<GGML_TYPE_Q4_1, block_q4_1>(pool, args, stream);
        break;
    case GGML_TYPE_Q5_0:
        mul_mat_q_case<GGML_TYPE_Q5_0, block_q5_0>(pool, args, stream);
        break;
    case GGML_TYPE_Q5_1:
        mul_mat_q_case<GGML_TYPE_Q5_1, block_q5_1>(pool, args, stream);
        break;
    case GGML_TYPE_Q8_0:
        mul_mat_q_case<GGML_TYPE_Q8_0, block_q8_0>(pool, args, stream);
        break;
    case GGML_TYPE_MXFP4:
        mul_mat_q_case<GGML_TYPE_MXFP4, block_mxfp4>(pool, args, stream);
        break;
    case GGML_TYPE_Q2_K:
        mul_mat_q_case<GGML_TYPE_Q2_K, block_q2_K>(pool, args, stream);
        break;
    case GGML_TYPE_Q3_K:
        mul_mat_q_case<GGML_TYPE_Q3_K, block_q3_K>(pool, args, stream);
        break;
    case GGML_TYPE_Q4_K:
        mul_mat_q_case<GGML_TYPE_Q4_K, block_q4_K>(pool, args, stream);
        break;
    case GGML_TYPE_Q5_K:
        mul_mat_q_case<GGML_TYPE_Q5_K, block_q5_K>(pool, args, stream);
        break;
    case GGML_TYPE_Q6_K:
        mul_mat_q_case<GGML_TYPE_Q6_K, block_q6_K>(pool, args, stream);
        break;
    case GGML_TYPE_IQ2_XXS:
        mul_mat_q_case<GGML_TYPE_IQ2_XXS, block_iq2_xxs>(pool, args, stream);
        break;
    case GGML_TYPE_IQ2_XS:
        mul_mat_q_case<GGML_TYPE_IQ2_XS, block_iq2_xs>(pool, args, stream);
        break;
    case GGML_TYPE_IQ2_S:
        mul_mat_q_case<GGML_TYPE_IQ2_S, block_iq2_s>(pool, args, stream);
        break;
    case GGML_TYPE_IQ3_XXS:
        mul_mat_q_case<GGML_TYPE_IQ3_XXS, block_iq3_xxs>(pool, args, stream);
        break;
    case GGML_TYPE_IQ3_S:
        mul_mat_q_case<GGML_TYPE_IQ3_S, block_iq3_s>(pool, args, stream);
        break;
    case GGML_TYPE_IQ1_S:
        mul_mat_q_case<GGML_TYPE_IQ1_S, block_iq1_s>(pool, args, stream);
        break;
    case GGML_TYPE_IQ4_XS:
        mul_mat_q_case<GGML_TYPE_IQ4_XS, block_iq4_xs>(pool, args, stream);
        break;
    case GGML_TYPE_IQ4_NL:
        mul_mat_q_case<GGML_TYPE_IQ4_NL, block_iq4_nl>(pool, args, stream);
        break;
    default:
        GGML_ABORT("fatal error");
        break;
    }
}
