#include "hip/hip_runtime.h"
#include <float.h>
#include <algorithm>
#include "common.cuh"
#include "cuda_func.h"

static __global__ void argmax_f32(const float* __restrict__ x, int32_t* __restrict__ dst, const int64_t ncols) {
    const int64_t row = blockIdx.x;

    float maxval = -FLT_MAX;
    int   argmax = -1;
    const float* rowx = x + row * ncols;

    for (int32_t col = threadIdx.x; col < ncols; col += blockDim.x) {
        const float val = rowx[col];
        if (val > maxval) {
            maxval = val;
            argmax = col;
        }
    }

#pragma unroll
    for (int offset = 16; offset > 0; offset >>= 1) {
        const float val = __shfl_xor_sync(0xFFFFFFFF, maxval, offset, WARP_SIZE);
        const int   col = __shfl_xor_sync(0xFFFFFFFF, argmax, offset, WARP_SIZE);
        if (val > maxval) {
            maxval = val;
            argmax = col;
        }
    }

    const int n_warps = blockDim.x / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;
    const int warp_id = threadIdx.x / WARP_SIZE;
    if (n_warps > 1) {
        constexpr int    max_warps = 1024 / WARP_SIZE;
        __shared__ float shared_maxval[max_warps];
        __shared__ int   shared_argmax[max_warps];
        if (lane_id == 0) {
            shared_maxval[warp_id] = maxval;
            shared_argmax[warp_id] = argmax;
        }

        __syncthreads();

        if (warp_id == 0) {
            if (lane_id < n_warps) {
                maxval = shared_maxval[lane_id];
                argmax = shared_argmax[lane_id];
            }
#pragma unroll
            for (int offset = 16; offset > 0; offset >>= 1) {
                const float val = __shfl_xor_sync(0xFFFFFFFF, maxval, offset, WARP_SIZE);
                const int   col = __shfl_xor_sync(0xFFFFFFFF, argmax, offset, WARP_SIZE);
                if (val > maxval) {
                    maxval = val;
                    argmax = col;
                }
            }
        }
    }

    if (warp_id == 0 && lane_id == 0) {
        dst[row] = argmax;
    }
}

void argmax_cuda(const argmax_context* ctx, hipStream_t stream)
{
    const int64_t num_blocks = ctx->nrows;
    const int64_t num_threads = std::min<int64_t>(1024, (ctx->ne00 + WARP_SIZE - 1) / WARP_SIZE * WARP_SIZE);
    const dim3 blocks_dim(num_threads, 1, 1);
    const dim3 blocks_num(num_blocks, 1, 1);

    argmax_f32 << <blocks_num, blocks_dim, 0, stream >> > (ctx->src0_d, ctx->dst_d, ctx->ne00);
}
