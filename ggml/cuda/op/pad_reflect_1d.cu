#include "hip/hip_runtime.h"
#include "common.cuh"

static constexpr size_t CUDA_PAD_REFLECT_1D_BLOCK_SIZE = 256;

static __global__ __launch_bounds__(CUDA_PAD_REFLECT_1D_BLOCK_SIZE, 1) void
pad_reflect_1d_kernel_f32(
    const void* __restrict__ src0,
    void* __restrict__       dst,
    const int64_t             ne0,
    const int64_t             ne00,
    const uint3               ne01,
    const int64_t             ne02,
    const int64_t             ne03,
    const int64_t             nb00,
    const int64_t             nb01,
    const int64_t             nb02,
    const int64_t             nb03,
    const int64_t             nb0,
    const int64_t             nb1,
    const int64_t             nb2,
    const int64_t             nb3,
    const int                 p0,
    const int                 /*p1*/) {
    const int64_t i3 = blockIdx.z;
    const int64_t i2 = blockIdx.y;

    const uint2   div_mod_packed = fast_div_modulo(blockIdx.x, ne01);
    const int64_t tile1 = div_mod_packed.y;  // i1
    const int64_t tile0 = div_mod_packed.x;  // nth i0 tile
    const int64_t i1 = tile1;
    const int64_t i0 = threadIdx.x + tile0 * blockDim.x;

    // ne01.z is original value of unpacked ne01 (see init_fastdiv_values in common.cuh)
    if (i0 >= ne0 || i1 >= ne01.z || i2 >= ne02 || i3 >= ne03) {
        return;
    }

    const char* src0_ptr = (const char*)src0 + i3 * nb03 + i2 * nb02 + i1 * nb01;
    char* dst_ptr = (char*)dst + i3 * nb3 + i2 * nb2 + i1 * nb1;

    const int64_t rel_i0 = i0 - p0;  // relative i0 in src0
    int64_t src_idx;

    if (rel_i0 < 0) {
        // Left padding - reflect
        src_idx = -rel_i0;
    }
    else if (rel_i0 < ne00) {
        // Middle - copy
        src_idx = rel_i0;
    }
    else {
        // Right padding - reflect
        src_idx = 2 * ne00 - 2 - rel_i0;
    }
    const float value = *(const float*)(src0_ptr + src_idx * nb00);
    *(float*)(dst_ptr + i0 * nb0) = value;
}

void pad_reflect_1d_cuda(
    const void* src0, void* dst,
    const int64_t ne0,
    const int64_t ne00, const int64_t ne01, const int64_t ne02, const int64_t ne03,
    const int64_t nb00, const int64_t nb01, const int64_t nb02, const int64_t nb03,
    const int64_t nb0, const int64_t nb1, const int64_t nb2, const int64_t nb3,
    const int p0, const int p1,hipStream_t stream)
{
    constexpr int64_t bx = CUDA_PAD_REFLECT_1D_BLOCK_SIZE;  // threads per block (x)

    const uint3   ne01_packed = init_fastdiv_values(ne01);
    const int64_t     tiles0 = (ne0 + bx - 1) / bx;             // number of tiles along i0
    // grid.x covers i1 and all tiles of i0: [ne01 * tiles0]
    // grid.y covers i2: [ne02]
    // grid.z covers i3: [ne03]
    const dim3        grid_dims((unsigned)(ne01 * tiles0), (unsigned)ne02, (unsigned)ne03);
    const dim3        block_dims((unsigned)bx, 1, 1);

    pad_reflect_1d_kernel_f32 << <grid_dims, block_dims, 0, stream >> > (
        src0, dst, ne0, ne00, ne01_packed, ne02, ne03, nb00, nb01, nb02, nb03,
        nb0, nb1, nb2, nb3, p0, p1);
}