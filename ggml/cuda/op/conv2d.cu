#include "hip/hip_runtime.h"
#include "cuda_func.h"
#include "internal_ds.h"
#include "convert.cuh"

struct conv_params {
    const int64_t IW, IH;
    const int64_t OW, OH;
    const int64_t KW, KH;
    const int64_t ST_X, ST_Y;
    const int64_t PD_X, PD_Y;
    const int64_t DL_X, DL_Y;
    const int64_t IC, OC;
    const int64_t B;
    const int64_t TOTAL;
};

struct kernel_bounds {
    int64_t y_min, y_max;
    int64_t x_min, x_max;
};

__device__ __forceinline__ int64_t max64(int64_t a, int64_t b) {
    return (a > b) ? a : b;
}

__device__ __forceinline__ int64_t min64(int64_t a, int64_t b) {
    return (a < b) ? a : b;
}

__device__ __forceinline__ kernel_bounds calculate_kernel_bounds(int64_t out_x, int64_t out_y, const conv_params& P) {
    kernel_bounds bounds;
    bounds.y_min = max64(0, (P.PD_Y - out_y * P.ST_Y + P.DL_Y - 1) / P.DL_Y);
    bounds.y_max = min64(P.KH, (P.IH + P.PD_Y - out_y * P.ST_Y + P.DL_Y - 1) / P.DL_Y);
    bounds.x_min = max64(0, (P.PD_X - out_x * P.ST_X + P.DL_X - 1) / P.DL_X);
    bounds.x_max = min64(P.KW, (P.IW + P.PD_X - out_x * P.ST_X + P.DL_X - 1) / P.DL_X);
    return bounds;
}

__device__ __forceinline__ int calculate_input_coord(int64_t out_coord,
    int64_t kern_coord,
    int64_t stride,
    int64_t dilation,
    int64_t padding) {
    return out_coord * stride + kern_coord * dilation - padding;
}

struct whcn_layout {
    __device__ static int64_t input_index(int64_t n, int64_t c, int64_t y, int64_t x, const conv_params& P) {
        return n * (P.IC * P.IW * P.IH) + c * P.IW * P.IH + y * P.IW + x;
    }

    __device__ static int64_t kernel_index(int64_t c_out, int64_t c_in, int64_t ky, int64_t kx, const conv_params& P) {
        return c_out * (P.IC * P.KH * P.KW) + c_in * (P.KH * P.KW) + ky * P.KW + kx;
    }

    __device__ static int64_t output_index(int64_t n, int64_t c, int64_t y, int64_t x, const conv_params& P) {
        return n * (P.OC * P.OW * P.OH) + c * P.OW * P.OH + y * P.OW + x;
    }

    __device__ static void unpack_indices(int64_t             global_idx,
        const conv_params& P,
        int64_t& n,
        int64_t& c,
        int64_t& out_y,
        int64_t& out_x) {
        out_x = global_idx % P.OW;
        out_y = (global_idx / P.OW) % P.OH;
        c = (global_idx / (P.OW * P.OH)) % P.OC;
        n = global_idx / (P.OW * P.OH * P.OC);
    }
};

template <typename T, typename Layout>
static __global__ void conv2d_kernel(const float* __restrict__ input,
    const T* __restrict__ kernel,
    float* __restrict__ output,
    const conv_params P) {
    const int64_t global_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_idx >= P.TOTAL) {
        return;
    }

    int64_t n, c_out, out_y, out_x;
    Layout::unpack_indices(global_idx, P, n, c_out, out_y, out_x);

    float acc = 0.0f;

    for (int64_t c_in = 0; c_in < P.IC; ++c_in) {
        kernel_bounds bounds = calculate_kernel_bounds(out_x, out_y, P);

        for (int64_t ky = bounds.y_min; ky < bounds.y_max; ++ky) {
            const int64_t in_y = calculate_input_coord(out_y, ky, P.ST_Y, P.DL_Y, P.PD_Y);

            for (int64_t kx = bounds.x_min; kx < bounds.x_max; ++kx) {
                const int64_t in_x = calculate_input_coord(out_x, kx, P.ST_X, P.DL_X, P.PD_X);

                const float input_val = input[Layout::input_index(n, c_in, in_y, in_x, P)];
                const T kernel_val = kernel[Layout::kernel_index(c_out, c_in, ky, kx, P)];
                acc += (input_val * ggml_cuda_cast<float>(kernel_val));
            }
        }
    }

    // [N, OC, OH, OW]
    output[Layout::output_index(n, c_out, out_y, out_x, P)] = acc;
}

template <typename T>
static void conv2d_cuda(const float* X_D, const T* K_D, float* Y_D, const conv_params P, hipStream_t st) {
    static constexpr size_t CUDA_CONV2D_BLOCK_SIZE = 256;
    const int blocks = (P.TOTAL + CUDA_CONV2D_BLOCK_SIZE - 1) / CUDA_CONV2D_BLOCK_SIZE;
    conv2d_kernel<T, whcn_layout> << <blocks, CUDA_CONV2D_BLOCK_SIZE, 0, st >> > (X_D, K_D, Y_D, P);
}

void conv2d_cuda(ggml_type kernel_type,
    const float* X_D, void* K_D, float* Y_D,
    const int IW, const int IH, const int OW, const int OH,
    const int KW, const int KH, const int ST_X, const int ST_Y,
    const int PD_X, const int PD_Y, const int DL_X, const int DL_Y,
    const int IC, const int OC, const int B, hipStream_t stream)
{
    const int64_t total = B * OC * OH * OW;
    conv_params   params = { IW, IH, OW, OH, KW, KH, ST_X, ST_Y, PD_X, PD_Y, DL_X, DL_Y, IC, OC, B, total };

    if (kernel_type == GGML_TYPE_F16) {
        conv2d_cuda(X_D, (half*)K_D, Y_D, params, stream);
    }
    else {
        conv2d_cuda(X_D, (float*)K_D, Y_D, params, stream);
    }
}