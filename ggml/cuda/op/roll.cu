#include "hip/hip_runtime.h"
static __forceinline__ __device__ int64_t wrap_index(const int64_t idx, const int64_t ne) {
    if (idx < 0) {
        return idx + ne;
    }
    if (idx >= ne) {
        return idx - ne;
    }
    return idx;
}

static __global__ void roll_f32_cuda(const float* __restrict__ src,
    float* __restrict__ dst,
    const int64_t ne00,
    const int64_t ne01,
    const int64_t ne02,
    const int64_t ne03,
    const int     s0,
    const int     s1,
    const int     s2,
    const int     s3) {
    const int64_t idx = int64_t(blockDim.x) * blockIdx.x + threadIdx.x;
    const int64_t n_elements = ne00 * ne01 * ne02 * ne03;

    if (idx >= n_elements) {
        return;
    }

    const int64_t i0 = idx % ne00;
    const int64_t i1 = (idx / ne00) % ne01;
    const int64_t i2 = (idx / (ne00 * ne01)) % ne02;
    const int64_t i3 = (idx / (ne00 * ne01 * ne02)) % ne03;

    const int64_t d0 = wrap_index(i0 - s0, ne00);
    const int64_t d1 = wrap_index(i1 - s1, ne01);
    const int64_t d2 = wrap_index(i2 - s2, ne02);
    const int64_t d3 = wrap_index(i3 - s3, ne03);

    dst[i3 * (ne00 * ne01 * ne02) + i2 * (ne01 * ne00) + i1 * ne00 + i0] =
        src[d3 * (ne00 * ne01 * ne02) + d2 * (ne01 * ne00) + d1 * ne00 + d0];
}

void roll_f32_cuda(const float* __restrict__ src,
    float* __restrict__ dst,
    const int64_t ne00,
    const int64_t ne01,
    const int64_t ne02,
    const int64_t ne03,
    const int     s0,
    const int     s1,
    const int     s2,
    const int     s3,
    hipStream_t stream)
{
    static constexpr size_t CUDA_ROLL_BLOCK_SIZE = 256;

    int64_t sz = (ne00 * ne01 * ne02 * ne03);
    int64_t num_blocks = (sz + CUDA_ROLL_BLOCK_SIZE - 1) / CUDA_ROLL_BLOCK_SIZE;

    roll_f32_cuda << <num_blocks, CUDA_ROLL_BLOCK_SIZE, 0, stream >> > (
        src, dst, ne00, ne01, ne02, ne03, s0, s1, s2, s3);
}