#include "hip/hip_runtime.h"
#include <algorithm>
#include "internal_ds.h"
#include "common.cuh"
#define GGML_ASSERT(...)
#define GGML_ABORT(...)

template <ggml_sort_order order>
static __global__ void k_argsort_f32_i32(const float* x, int* dst, const int ncols, int ncols_pad) {
    // bitonic sort
    int col = threadIdx.x;
    int row = blockIdx.y;

    if (col >= ncols_pad) {
        return;
    }

    const float* x_row = x + row * ncols;
    extern __shared__ int dst_row[];

    // initialize indices
    dst_row[col] = col;

    __syncthreads();

    for (int k = 2; k <= ncols_pad; k *= 2) {
        for (int j = k / 2; j > 0; j /= 2) {
            int ixj = col ^ j;
            if (ixj > col) {
                if ((col & k) == 0) {
                    if (dst_row[col] >= ncols ||
                        (dst_row[ixj] < ncols && (order == GGML_SORT_ORDER_ASC ?
                            x_row[dst_row[col]] > x_row[dst_row[ixj]] :
                    x_row[dst_row[col]] < x_row[dst_row[ixj]]))
                        ) {
                        std::swap(dst_row[col], dst_row[ixj]);
                    }
                }
                else {
                    if (dst_row[ixj] >= ncols ||
                        (dst_row[col] < ncols && (order == GGML_SORT_ORDER_ASC ?
                            x_row[dst_row[col]] < x_row[dst_row[ixj]] :
                            x_row[dst_row[col]] > x_row[dst_row[ixj]]))
                        ) {
                        std::swap(dst_row[col], dst_row[ixj]);
                    }
                }
            }
            __syncthreads();
        }
    }

    // copy the result to dst without the padding
    if (col < ncols) {
        dst[row * ncols + col] = dst_row[col];
    }
}

static int next_power_of_2(int x) {
    int n = 1;
    while (n < x) {
        n *= 2;
    }
    return n;
}

void argsort_f32_i32_cuda(
    const float* x, int* dst,
    const int ncols, const int nrows,
    ggml_sort_order order, hipStream_t stream)
{
    // bitonic sort requires ncols to be power of 2
    const int ncols_pad = next_power_of_2(ncols);

    const dim3 block_dims(ncols_pad, 1, 1);
    const dim3 block_nums(1, nrows, 1);
    const size_t shared_mem = ncols_pad * sizeof(int);

    // FIXME: this limit could be raised by ~2-4x on Ampere or newer
    GGML_ASSERT(shared_mem <= ggml_cuda_info().devices[ggml_cuda_get_device()].smpb);

    if (order == GGML_SORT_ORDER_ASC) {
        k_argsort_f32_i32<GGML_SORT_ORDER_ASC> << <block_nums, block_dims, shared_mem, stream >> > (x, dst, ncols, ncols_pad);
    }
    else if (order == GGML_SORT_ORDER_DESC) {
        k_argsort_f32_i32<GGML_SORT_ORDER_DESC> << <block_nums, block_dims, shared_mem, stream >> > (x, dst, ncols, ncols_pad);
    }
    else {
        GGML_ABORT("fatal error");
    }
}