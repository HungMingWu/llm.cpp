#include "hip/hip_runtime.h"
#include "common.cuh"
template <int block_size>
static __global__ void norm_f32(
    const float* x, float* dst, const int ncols, const int64_t stride_row, const int64_t stride_channel,
    const int64_t stride_sample, const float eps) {
    const int nrows = gridDim.x;
    const int nchannels = gridDim.y;

    const int row = blockIdx.x;
    const int channel = blockIdx.y;
    const int sample = blockIdx.z;
    const int tid = threadIdx.x;

    x += sample * stride_sample + channel * stride_channel + row * stride_row;
    dst += ((sample * nchannels + channel) * nrows + row) * ncols;

    float2 mean_var = make_float2(0.0f, 0.0f);

    for (int col = tid; col < ncols; col += block_size) {
        const float xi = x[col];
        mean_var.x += xi;
        mean_var.y += xi * xi;
    }

    // sum up partial sums
    mean_var = warp_reduce_sum(mean_var);
    if constexpr (block_size > WARP_SIZE) {
        static_assert(block_size == 1024, "unexpected block_size");
        __shared__ float2 s_sum[32];
        const int warp_id = threadIdx.x / WARP_SIZE;
        const int lane_id = threadIdx.x % WARP_SIZE;
        if (lane_id == 0) {
            s_sum[warp_id] = mean_var;
        }
        __syncthreads();
        mean_var = s_sum[lane_id];
        mean_var = warp_reduce_sum(mean_var);
    }

    const float mean = mean_var.x / ncols;
    const float var = mean_var.y / ncols - mean * mean;
    const float inv_std = rsqrtf(var + eps);

    for (int col = tid; col < ncols; col += block_size) {
        dst[col] = (x[col] - mean) * inv_std;
    }
}

void norm_f32_cuda(
    const float* x, float* dst, const int ncols, const int nrows, const int nchannels, const int nsamples,
    const int64_t stride_row, const int64_t stride_channel, const int64_t stride_sample, const float eps, hipStream_t stream)
{
    const dim3 blocks_num(nrows, nchannels, nsamples);
    if (ncols < 1024) {
        const dim3 block_dims(WARP_SIZE, 1, 1);
        norm_f32<WARP_SIZE> << <blocks_num, block_dims, 0, stream >> > (x, dst, ncols, stride_row, stride_channel, stride_sample, eps);
    }
    else {
        const dim3 block_dims(1024, 1, 1);
        norm_f32<1024> << <blocks_num, block_dims, 0, stream >> > (x, dst, ncols, stride_row, stride_channel, stride_sample, eps);
    }
}

template <int block_size, bool do_multiply = false>
static __global__ void rms_norm_f32(
    const float* x, float* dst, const int ncols, const int64_t stride_row, const int64_t stride_channel,
    const int64_t stride_sample, const float eps, const float* mul = nullptr, const int64_t mul_stride_row = 0,
    const int64_t mul_stride_channel = 0, const int64_t mul_stride_sample = 0, const int mul_ncols = 0,
    const int mul_nrows = 0, const int mul_nchannels = 0, const int mul_nsamples = 0) {
    const int nrows = gridDim.x;
    const int nchannels = gridDim.y;

    const int row = blockIdx.x;
    const int channel = blockIdx.y;
    const int sample = blockIdx.z;
    const int tid = threadIdx.x;

    x += sample * stride_sample + channel * stride_channel + row * stride_row;
    dst += ((sample * nchannels + channel) * nrows + row) * ncols;

    if constexpr (do_multiply) {
        const int mul_row = row % mul_nrows;
        const int mul_channel = channel % mul_nchannels;
        const int mul_sample = sample % mul_nsamples;
        mul += mul_sample * mul_stride_sample + mul_channel * mul_stride_channel + mul_row * mul_stride_row;
    }

    float tmp = 0.0f; // partial sum for thread in warp

    for (int col = tid; col < ncols; col += block_size) {
        const float xi = x[col];
        tmp += xi * xi;
    }

    // sum up partial sums
    tmp = warp_reduce_sum(tmp);
    if constexpr (block_size > WARP_SIZE) {
        static_assert(block_size == 1024, "unexpected block_size");
        __shared__ float s_sum[32];
        const int warp_id = threadIdx.x / WARP_SIZE;
        const int lane_id = threadIdx.x % WARP_SIZE;
        if (lane_id == 0) {
            s_sum[warp_id] = tmp;
        }
        __syncthreads();
        tmp = s_sum[lane_id];
        tmp = warp_reduce_sum(tmp);
    }

    const float mean = tmp / ncols;
    const float scale = rsqrtf(mean + eps);

    for (int col = tid; col < ncols; col += block_size) {
        if constexpr (do_multiply) {
            const int mul_col = col % mul_ncols;
            dst[col] = scale * x[col] * mul[mul_col];
        }
        else {
            dst[col] = scale * x[col];
        }
    }
}

void rms_norm_f32_cuda(
    const float* x, float* dst, const int ncols, const int nrows, const int nchannels, const int nsamples,
    const int64_t stride_row, const int64_t stride_channel, const int64_t stride_sample, const float eps, hipStream_t stream) {
    const dim3 blocks_num(nrows, nchannels, nsamples);
    if (ncols < 1024) {
        const dim3 block_dims(WARP_SIZE, 1, 1);
        rms_norm_f32<WARP_SIZE, false> << <blocks_num, block_dims, 0, stream >> > (x, dst, ncols, stride_row, stride_channel, stride_sample, eps);
    }
    else {
        const dim3 block_dims(1024, 1, 1);
        rms_norm_f32<1024, false> << <blocks_num, block_dims, 0, stream >> > (x, dst, ncols, stride_row, stride_channel, stride_sample, eps);
    }
}

template <int block_size>
static __global__ void rms_norm_back_f32(
    const float* grad, const float* xf, float* dst, const int ncols, const float eps) {
    const int row = blockIdx.x * blockDim.y + threadIdx.y;
    const int tid = threadIdx.x;

    grad += int64_t(row) * ncols;
    xf += int64_t(row) * ncols;
    dst += int64_t(row) * ncols;

    float sum_xx = 0.0f; // sum for squares of x, equivalent to forward pass
    float sum_xg = 0.0f; // sum for x * gradient, needed because RMS norm mixes inputs

    for (int col = tid; col < ncols; col += block_size) {
        const float xfi = xf[col];
        sum_xx += xfi * xfi;
        sum_xg += xfi * grad[col];
    }

    // sum up partial sums
    sum_xx = warp_reduce_sum(sum_xx);
    sum_xg = warp_reduce_sum(sum_xg);
    if constexpr (block_size > WARP_SIZE) {
        static_assert(block_size == 1024, "unexpected block_size");
        __shared__ float s_sum_xx[32];
        __shared__ float s_sum_xg[32];
        const int warp_id = threadIdx.x / WARP_SIZE;
        const int lane_id = threadIdx.x % WARP_SIZE;
        if (lane_id == 0) {
            s_sum_xx[warp_id] = sum_xx;
            s_sum_xg[warp_id] = sum_xg;
        }
        __syncthreads();

        sum_xx = s_sum_xx[lane_id];
        sum_xx = warp_reduce_sum(sum_xx);

        sum_xg = s_sum_xg[lane_id];
        sum_xg = warp_reduce_sum(sum_xg);
    }

    const float mean_eps = sum_xx / ncols + eps;
    const float sum_eps = sum_xx + ncols * eps;

    const float scale_grad = rsqrtf(mean_eps);
    const float scale_x = -scale_grad * sum_xg / sum_eps;

    for (int col = tid; col < ncols; col += block_size) {
        dst[col] = scale_grad * grad[col] + scale_x * xf[col];
    }
}

void rms_norm_back_f32_cuda(const float* grad, const float* xf, float* dst, const int ncols, const int nrows, const float eps, hipStream_t stream) {
    if (ncols < 1024) {
        const dim3 block_dims(WARP_SIZE, 1, 1);
        rms_norm_back_f32<WARP_SIZE> << <nrows, block_dims, 0, stream >> > (grad, xf, dst, ncols, eps);
    }
    else {
        const dim3 block_dims(1024, 1, 1);
        rms_norm_back_f32<1024> << <nrows, block_dims, 0, stream >> > (grad, xf, dst, ncols, eps);
    }
}

template <int block_size>
static __global__ void group_norm_f32(const float* x, float* dst, const int group_size, const int ne_elements, const float eps) {
    // blockIdx.x: num_groups idx
    // threadIdx.x: block_size idx
    const int start = blockIdx.x * group_size + threadIdx.x;
    const int end = min(blockIdx.x * group_size + group_size, ne_elements);

    float tmp = 0.0f; // partial sum for thread in warp

    for (int j = start; j < end; j += block_size) {
        tmp += x[j];
    }

    tmp = warp_reduce_sum(tmp);
    if constexpr (block_size > WARP_SIZE) {
        static_assert(block_size == 1024, "unexpected block_size");
        __shared__ float s_sum[32];
        const int warp_id = threadIdx.x / WARP_SIZE;
        const int lane_id = threadIdx.x % WARP_SIZE;
        if (lane_id == 0) {
            s_sum[warp_id] = tmp;
        }
        __syncthreads();
        tmp = s_sum[lane_id];
        tmp = warp_reduce_sum(tmp);
    }

    const float mean = tmp / group_size;
    tmp = 0.0f;

    for (int j = start; j < end; j += block_size) {
        const float xi = x[j] - mean;
        dst[j] = xi;
        tmp += xi * xi;
    }

    tmp = warp_reduce_sum(tmp);
    if (block_size > WARP_SIZE) {
        __shared__ float s_sum[32];
        const int warp_id = threadIdx.x / WARP_SIZE;
        const int lane_id = threadIdx.x % WARP_SIZE;
        if (lane_id == 0) {
            s_sum[warp_id] = tmp;
        }
        __syncthreads();
        tmp = s_sum[lane_id];
        tmp = warp_reduce_sum(tmp);
    }

    const float variance = tmp / group_size;
    const float scale = rsqrtf(variance + eps);
    for (int j = start; j < end; j += block_size) {
        dst[j] *= scale;
    }
}

void group_norm_f32_cuda(
    const float* x, float* dst, const int num_groups, const float eps, const int group_size, const int ne_elements, hipStream_t stream) {
    if (group_size < 1024) {
        const dim3 block_dims(WARP_SIZE, 1, 1);
        group_norm_f32<WARP_SIZE> << <num_groups, block_dims, 0, stream >> > (x, dst, group_size, ne_elements, eps);
    }
    else {
        const dim3 block_dims(1024, 1, 1);
        group_norm_f32<1024> << <num_groups, block_dims, 0, stream >> > (x, dst, group_size, ne_elements, eps);
    }
}

// template <int block_size>
// static __global__ void l2_norm_f32(const float * x, float * dst, const int ncols, const float eps) {
//     const int row = blockIdx.x*blockDim.y + threadIdx.y;
//     const int tid = threadIdx.x;

//     float tmp = 0.0f; // partial sum for thread in warp

//     for (int col = tid; col < ncols; col += block_size) {
//         const float xi = x[row*ncols + col];
//         tmp += xi * xi;
//     }

//     // sum up partial sums
//     tmp = warp_reduce_sum(tmp);
//     if (block_size > WARP_SIZE) {
//         __shared__ float s_sum[32];
//         int warp_id = threadIdx.x / WARP_SIZE;
//         int lane_id = threadIdx.x % WARP_SIZE;
//         if (lane_id == 0) {
//             s_sum[warp_id] = tmp;
//         }
//         __syncthreads();
//         tmp = s_sum[lane_id];
//         tmp = warp_reduce_sum(tmp);
//     }

//     // from https://pytorch.org/docs/stable/generated/torch.nn.functional.normalize.html
//     const float scale = rsqrtf(fmaxf(tmp, eps * eps));

//     for (int col = tid; col < ncols; col += block_size) {
//         dst[row*ncols + col] = scale * x[row*ncols + col];
//     }
// }

template <int block_size>
static __global__ void l2_norm_f32(
    const float* x, float* dst, const int ncols, const int64_t stride_row, const int64_t stride_channel,
    const int64_t stride_sample, const float eps) {
    const int nrows = gridDim.x;
    const int nchannels = gridDim.y;

    const int row = blockIdx.x;
    const int channel = blockIdx.y;
    const int sample = blockIdx.z;
    const int tid = threadIdx.x;

    x += sample * stride_sample + channel * stride_channel + row * stride_row;
    dst += ((sample * nchannels + channel) * nrows + row) * ncols;

    float tmp = 0.0f; // partial sum for thread in warp

    for (int col = tid; col < ncols; col += block_size) {
        const float xi = x[col];
        tmp += xi * xi;
    }

    // sum up partial sums
    tmp = warp_reduce_sum(tmp);
    if constexpr (block_size > WARP_SIZE) {
        static_assert(block_size == 1024, "unexpected block_size");
        __shared__ float s_sum[32];
        const int warp_id = threadIdx.x / WARP_SIZE;
        const int lane_id = threadIdx.x % WARP_SIZE;
        if (lane_id == 0) {
            s_sum[warp_id] = tmp;
        }
        __syncthreads();
        tmp = s_sum[lane_id];
        tmp = warp_reduce_sum(tmp);
    }

    // from https://pytorch.org/docs/stable/generated/torch.nn.functional.normalize.html
    const float scale = rsqrtf(fmaxf(tmp, eps * eps));

    for (int col = tid; col < ncols; col += block_size) {
        dst[col] = scale * x[col];
    }
}

void l2_norm_f32_cuda(
    const float* x, float* dst, const int ncols, const int nrows, const int nchannels, const int nsamples,
    const int64_t stride_row, const int64_t stride_channel,
    const int64_t stride_sample, const float eps, hipStream_t stream)
{
    const dim3 blocks_num(nrows, nchannels, nsamples);
    if (ncols < 1024) {
        const dim3 block_dims(WARP_SIZE, 1, 1);
        l2_norm_f32<WARP_SIZE> << <blocks_num, block_dims, 0, stream >> > (x, dst, ncols, stride_row, stride_channel, stride_sample, eps);
    }
    else {
        const dim3 block_dims(1024, 1, 1);
        l2_norm_f32<1024> << <blocks_num, block_dims, 0, stream >> > (x, dst, ncols, stride_row, stride_channel, stride_sample, eps);
    }
}

void rms_norm_mul_f32_cuda(
    const float* x, const float* mul, float* dst, const int ncols, const int nrows, const int nchannels, const int nsamples,
    const int64_t stride_row, const int64_t stride_channel, const int64_t stride_sample,
    const int64_t mul_stride_row, const int64_t mul_stride_channel, const int64_t mul_stride_sample,
    const int mul_ncols, const int mul_nrows, const int mul_nchannels, const int mul_nsamples,
    const float eps, hipStream_t stream)
{
    const dim3 blocks_num(nrows, nchannels, nsamples);
    if (mul == nullptr) {
        rms_norm_f32_cuda(x, dst, ncols, nrows, nchannels, nsamples, stride_row, stride_channel, stride_sample, eps, stream);
        return;
    }
    if (ncols < 1024) {
        const dim3 block_dims(WARP_SIZE, 1, 1);
        rms_norm_f32<WARP_SIZE, true> << <blocks_num, block_dims, 0, stream >> > (x, dst, ncols, stride_row, stride_channel, stride_sample, eps, mul, mul_stride_row, mul_stride_channel, mul_stride_sample, mul_ncols, mul_nrows, mul_nchannels, mul_nsamples);
    }
    else {
        const dim3 block_dims(1024, 1, 1);
        rms_norm_f32<1024, true> << <blocks_num, block_dims, 0, stream >> > (x, dst, ncols, stride_row, stride_channel, stride_sample, eps, mul, mul_stride_row, mul_stride_channel, mul_stride_sample, mul_ncols, mul_nrows, mul_nchannels, mul_nsamples);
    }
}