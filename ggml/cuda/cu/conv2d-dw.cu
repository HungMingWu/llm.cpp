#include "hip/hip_runtime.h"
#include "cuda_func.h"
#define GGML_ABORT(...)

struct conv_params {
    int in_w, in_h;
    int out_w, out_h;
    int kernel_w, kernel_h;
    int stride_x, stride_y;
    int padding_x, padding_y;
    int dilation_x, dilation_y;
    int channels, batches;
};

struct kernel_bounds {
    int y_min, y_max;
    int x_min, x_max;
};

__device__ __forceinline__ kernel_bounds calculate_kernel_bounds(int out_x, int out_y, const conv_params& params) {
    kernel_bounds bounds;
    bounds.y_min = max(0, (params.padding_y - out_y * params.stride_y + params.dilation_y - 1) / params.dilation_y);
    bounds.y_max =
        min(params.kernel_h,
            (params.in_h + params.padding_y - out_y * params.stride_y + params.dilation_y - 1) / params.dilation_y);
    bounds.x_min = max(0, (params.padding_x - out_x * params.stride_x + params.dilation_x - 1) / params.dilation_x);
    bounds.x_max =
        min(params.kernel_w,
            (params.in_w + params.padding_x - out_x * params.stride_x + params.dilation_x - 1) / params.dilation_x);
    return bounds;
}

__device__ __forceinline__ int calculate_input_coord(int out_coord, int kern_coord, int stride, int dilation, int padding) {
    return out_coord * stride + kern_coord * dilation - padding;
}

struct whcn_layout {
    __device__ static int input_index(int n, int c, int y, int x, const conv_params& params) {
        return n * (params.channels * params.in_w * params.in_h) + c * params.in_w * params.in_h + y * params.in_w + x;
    }

    __device__ static int kernel_index(int c, int ky, int kx, const conv_params& params) {
        return c * params.kernel_h * params.kernel_w + ky * params.kernel_w + kx;
    }

    __device__ static int output_index(int n, int c, int y, int x, const conv_params& params) {
        return n * (params.channels * params.out_w * params.out_h) + c * params.out_w * params.out_h +
            y * params.out_w + x;
    }

    __device__ static void unpack_indices(int global_idx, const conv_params& params, int& n, int& c, int& out_y,
        int& out_x) {
        out_x = global_idx % params.out_w;
        out_y = (global_idx / params.out_w) % params.out_h;
        c = (global_idx / (params.out_w * params.out_h)) % params.channels;
        n = global_idx / (params.out_w * params.out_h * params.channels);
    }
};

struct cwhn_layout {
    __device__ static int input_index(int n, int c, int y, int x, const conv_params& params) {
        return n * (params.channels * params.in_w * params.in_h) + (y * params.in_w + x) * params.channels + c;
    }

    __device__ static int kernel_index(int c, int ky, int kx, const conv_params& params) {
        return (ky * params.kernel_w + kx) * params.channels + c;
    }

    __device__ static int output_index(int n, int c, int y, int x, const conv_params& params) {
        return n * (params.channels * params.out_w * params.out_h) + y * (params.out_w * params.channels) +
            x * params.channels + c;
    }

    __device__ static void unpack_indices(int global_idx, const conv_params& params, int& n, int& c, int& out_y,
        int& out_x) {
        c = global_idx % params.channels;
        out_x = (global_idx / params.channels) % params.out_w;
        out_y = (global_idx / (params.channels * params.out_w)) % params.out_h;
        n = global_idx / (params.channels * params.out_w * params.out_h);
    }
};

template <typename T, typename Layout>
__global__ void conv2d_dw_kernel(const T* __restrict__ input, const T* __restrict__ kernel, T* __restrict__ output,
    const int in_w, const int in_h, const int out_w, const int out_h,
    const int kernel_w, const int kernel_h, const int stride_x, const int stride_y,
    const int padding_x, const int padding_y, const int dilation_x, const int dilation_y,
    const int channels, const int batches) {
    const int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_elements = batches * channels * out_h * out_w;

    if (global_idx >= total_elements) {
        return;
    }

    conv_params params = { in_w,     in_h,      out_w,     out_h,      kernel_w,   kernel_h, stride_x,
                           stride_y, padding_x, padding_y, dilation_x, dilation_y, channels, batches };

    int batch_idx, channel_idx, out_y_idx, out_x_idx;
    Layout::unpack_indices(global_idx, params, batch_idx, channel_idx, out_y_idx, out_x_idx);

    T accumulator = 0;
    kernel_bounds bounds = calculate_kernel_bounds(out_x_idx, out_y_idx, params);

    for (int kern_y = bounds.y_min; kern_y < bounds.y_max; ++kern_y) {
        int in_y_idx = calculate_input_coord(out_y_idx, kern_y, params.stride_y, params.dilation_y, params.padding_y);

        for (int kern_x = bounds.x_min; kern_x < bounds.x_max; ++kern_x) {
            int in_x_idx = calculate_input_coord(out_x_idx, kern_x, params.stride_x, params.dilation_x, params.padding_x);

            const T input_val = input[Layout::input_index(batch_idx, channel_idx, in_y_idx, in_x_idx, params)];
            const T kernel_val = kernel[Layout::kernel_index(channel_idx, kern_y, kern_x, params)];

            accumulator += input_val * kernel_val;
        }
    }

    output[Layout::output_index(batch_idx, channel_idx, out_y_idx, out_x_idx, params)] = accumulator;
}

void conv2d_dw_cuda(conv2d_dw_context* ctx, hipStream_t stream)
{
    static constexpr size_t CUDA_CONV2D_DW_BLOCK_SIZE = 256;
    const int total = ctx->batches * ctx->channels * ctx->out_h * ctx->out_w;
    const int blocks = (total + CUDA_CONV2D_DW_BLOCK_SIZE - 1) / CUDA_CONV2D_DW_BLOCK_SIZE;

    if (ctx->input_is_contiguous) {
        conv2d_dw_kernel<float, whcn_layout> << <blocks, CUDA_CONV2D_DW_BLOCK_SIZE, 0, stream >> > (
            ctx->x_d, ctx->w_d, ctx->y_d,
            ctx->in_w, ctx->in_h, ctx->out_w, ctx->out_h,
            ctx->kernel_w, ctx->kernel_h, ctx->stride_x, ctx->stride_y, ctx->padding_x, ctx->padding_y,
            ctx->dilation_x, ctx->dilation_y, ctx->channels, ctx->batches);
    }
    else if (ctx->input_is_contiguous_channels) {
        conv2d_dw_kernel<float, cwhn_layout> << <blocks, CUDA_CONV2D_DW_BLOCK_SIZE, 0, stream >> > (
            ctx->x_d, ctx->w_d, ctx->y_d,
            ctx->in_w, ctx->in_h, ctx->out_w, ctx->out_h,
            ctx->kernel_w, ctx->kernel_h, ctx->stride_x, ctx->stride_y, ctx->padding_x, ctx->padding_y,
            ctx->dilation_x, ctx->dilation_y, ctx->channels, ctx->batches);
    }
    else {
        GGML_ABORT("Unsupported memory layout for conv_2d_dw");
    }
}